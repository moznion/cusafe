#include "cusafe.h"

void cudaSetDeviceSafe(int device_id) {
    hipError_t err = hipSetDevice(device_id);

    if (err) {
        fprintf(stderr, "Cannot set a device (device: %d)\n", device_id);
        exit(1);
    }
}

void cudaMallocSafe(void* ptr, int data_size) {
    hipError_t err = hipMalloc((void **)ptr, data_size);

    if (err) {
        fprintf(stderr, "Failed to allocate the memory on device\n");
        exit(1);
    }
}

void cudaMallocHostSafe(void* ptr, int data_size) {
    hipError_t err = hipHostMalloc((void **)ptr, data_size);

    if (err) {
        fprintf(stderr, "Failed to allocate the memory on host\n");
        exit(1);
    }
}

void cudaMemcpySafe(void* dst, const void* src, size_t data_size, hipMemcpyKind kind) {
    hipError_t err = hipMemcpy(dst, src, data_size, kind);

    if (err) {
        fprintf(stderr, "Failed to copy memory (direction: %s)\n", kind);
        exit(1);
    }
}

void cudaMemcpyPeerSafe(void* dst, int dst_device, const void* src, int src_device, size_t data_size) {
    hipError_t err = hipMemcpyPeer(dst, dst_device, src, src_device, data_size);

    if (err) {
        fprintf(stderr, "Failed to copy memory via peer (direction: %d -> %d)\n", src_device, dst_device);
        exit(1);
    }
}

void cudaFreeSafe(void* ptr) {
    hipError_t err = hipFree(ptr);

    if (err) {
        fprintf(stderr, "Failed to make device memory free\n");
        exit(1);
    }
}

void cudaFreeHostSafe(void* ptr) {
    hipError_t err = hipHostFree(ptr);

    if (err) {
        fprintf(stderr, "Failed to make host memory free\n");
        exit(1);
    }
}

